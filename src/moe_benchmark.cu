#include "hip/hip_runtime.h"
#include "common.h"

const int data_size = 128;
const int num_nodes = 16;
const int ppn = 8;
const int num_ranks = num_nodes * ppn;

int pipeline_parallelism(hipStream_t stream, ncclComm_t comm, int rank, int size, void *send_buf, void *recv_buf) {
    int peer = rank % 32 < 16 ? rank + 16 : rank - 16;

    NCCLCHECK(ncclSend((char *)send_buf, size, ncclFloat, peer, comm, stream));
    NCCLCHECK(ncclRecv((char *)recv_buf, size, ncclFloat, peer, comm, stream));
    return 0;
}

int experts_parallelism(hipStream_t stream, ncclComm_t comm, int rank, int size, void *send_buf, void *recv_buf) {

    int count = size / 4;
    int commFirstRank = 16*(rank / 16);
    int peer;

    for (int off=0; off < 16; off++) {
        peer = commFirstRank + off;
        NCCLCHECK(ncclSend(send_buf, count, ncclFloat, peer, comm, stream));
        NCCLCHECK(ncclRecv(recv_buf, count, ncclFloat, peer, comm, stream));
    }
    return 0;
}

testResult_t InitData(struct threadArgs* args, ncclDataType_t type, ncclRedOp_t op, int root, int rep, int in_place) {
  size_t sendcount = args->sendBytes / wordSize(type);
  size_t recvcount = args->expectedBytes / wordSize(type);
  int nranks = args->nProcs*args->nThreads*args->nGpus;

  for (int i=0; i<args->nGpus; i++) {
    CUDACHECK(hipSetDevice(args->gpus[i]));
    int rank = ((args->proc*args->nThreads + args->thread)*args->nGpus + i);
    CUDACHECK(hipMemset(args->recvbuffs[i], 0, args->expectedBytes));
    void* data = in_place ? args->recvbuffs[i] : args->sendbuffs[i];
    TESTCHECK(InitData(data, sendcount, 0, type, ncclSum, 33*rep + rank, 1, 0));
    for (int j=0; j<nranks; j++) {
      size_t partcount = sendcount/nranks;
      TESTCHECK(InitData((char*)args->expected[i] + j*partcount*wordSize(type), partcount, rank*partcount, type, ncclSum, 33*rep + j, 1, 0));
    }
    CUDACHECK(hipDeviceSynchronize());
  }
  // We don't support in-place alltoall
  args->reportErrors = in_place ? 0 : 1;
  return testSuccess;
}

void GetBw(size_t count, int typesize, double sec, double* algBw, double* busBw, int nranks) { // TODO
  double baseBw = (double)(count * nranks * typesize) / 1.0E9 / sec;

  *algBw = baseBw;
  double factor = ((double)(nranks-1))/((double)(nranks));
  *busBw = baseBw * factor;
}

void GetCollByteCount(size_t *sendcount, size_t *recvcount, size_t *paramcount, size_t *sendInplaceOffset, size_t *recvInplaceOffset, size_t count, int nranks) {
    *sendcount = (count/nranks)*nranks;
    *recvcount = (count/nranks)*nranks;
    *sendInplaceOffset = 0;
    *recvInplaceOffset = 0;
    *paramcount = count/nranks;
}

testResult_t RunColl(void* sendbuff, void* recvbuff, size_t count, ncclDataType_t type, ncclRedOp_t op, int root, ncclComm_t comm, hipStream_t stream) {
    /* The env var EXPERTS_REDUCTIONS_OP controls the collective simulating experts reductions
     * should be 0 for Reduce Scatter and 1 for All Gather
     * The env var PARALLEL_OP controls the additional coll, 0=experts parallelism, 1=pipeline parallelism
     */
    char *env;
    int rank, size, experts_op, parallel_op;
    ncclResult_t state;
    NCCLCHECK(ncclCommCount(comm, &size));
    NCCLCHECK(ncclCommUserRank(comm, &rank));

    env = getenv("EXPERTS_REDUCTIONS_OP");
    experts_op = env ? atoi(env) : 0;

    env = getenv("PARALLEL_OP");
    parallel_op = env ? atoi(env) : 0;

    if (size < 32){
        printf("This test is meant to be ran with at least 32 nodes.");
        return testNcclError;
    }

    ncclComm_t expertsComm;
    NCCLCHECK(ncclCommSplit(comm, rank % 32, 0, &expertsComm, NULL));
    do {
        NCCLCHECK(ncclCommGetAsyncError(comm, &state));
    } while(state == ncclInProgress);

    // Run desired scenario
    NCCLCHECK(ncclGroupStart());
    // Run experts RS/AG
    switch (experts_op){
        case 0:
            NCCLCHECK(ncclReduceScatter((char *)sendbuff, (char *) recvbuff, size, ncclFloat, ncclSum, expertsComm, stream));
            break;
        case 1:
            NCCLCHECK(ncclAllGather((char *)sendbuff, (char *) recvbuff, size/num_ranks, ncclFloat, expertsComm, stream));
            break;
        default:
            printf("Invalid experts_op value, should be 0 for RS or 1 for AG, but received %d\n", experts_op);
            return testNcclError;
    }

    // Run another collective in parallel
    switch (parallel_op){
        case 0:
            experts_parallelism(stream, comm, rank, size, sendbuff, recvbuff);
            break;
        case 1:
            pipeline_parallelism(stream, comm, rank, size, sendbuff, recvbuff);
            break;
        default:
            printf("Invalid parallel op value, should be 0 for experts parallelism and 1 for pipeline parallelism");
            return testNcclError;
    }
    
    NCCLCHECK(ncclGroupEnd());

    ncclCommDestroy(expertsComm);
    return testSuccess;
}


struct testColl moeBenchmarkTest = {
  "MoeBenchmark",
  GetCollByteCount,
  InitData,
  GetBw,
  RunColl
};

void GetBuffSize(size_t *sendcount, size_t *recvcount, size_t count, int nranks) {
  size_t paramcount, sendInplaceOffset, recvInplaceOffset;
  GetCollByteCount(sendcount, recvcount, &paramcount, &sendInplaceOffset, &recvInplaceOffset, count, nranks);
}

testResult_t RunTest(struct threadArgs* args, int root, ncclDataType_t type, const char* typeName, ncclRedOp_t op, const char* opName) {
  args->collTest = &moeBenchmarkTest;
  ncclDataType_t *run_types;
  const char **run_typenames;
  int type_count;

  if ((int)type != -1) {
    type_count = 1;
    run_types = &type;
    run_typenames = &typeName;
  } else {
    type_count = test_typenum;
    run_types = test_types;
    run_typenames = test_typenames;
  }

  for (int i=0; i<type_count; i++) {
      TESTCHECK(TimeTest(args, run_types[i], run_typenames[i], (ncclRedOp_t)0, "none", -1));
  }
  return testSuccess;
}

struct testEngine moeBenchmarkEngine = {
  GetBuffSize,
  RunTest
};

#pragma weak ncclTestEngine=moeBenchmarkEngine
