#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "common.h"
#include <algorithm>    // std::max


const int data_size = 128;
const int num_nodes = 16;
const int ppn = 8;
const int num_ranks = num_nodes * ppn;

static double parsesize(const char *value) {
    long long int units;
    double size;
    char size_lit;

    int count = sscanf(value, "%lf %1s", &size, &size_lit);

    switch (count) {
    case 2:
      switch (size_lit) {
      case 'G':
      case 'g':
        units = 1024*1024*1024;
        break;
      case 'M':
      case 'm':
        units = 1024*1024;
        break;
      case 'K':
      case 'k':
        units = 1024;
        break;
      default:
        return -1.0;
      };
      break;
    case 1:
      units = 1;
      break;
    default:
      return -1.0;
    }

    return size * units;
}

int pipeline_parallelism(hipStream_t stream, ncclComm_t comm, int rank, int count, void *send_buf, void *recv_buf) {
    int peer = rank % 32 < 16 ? rank + 16 : rank - 16;

    ncclGroupStart();
    NCCLCHECK(ncclSend((char *)send_buf, count, ncclChar, peer, comm, stream));
    NCCLCHECK(ncclRecv((char *)recv_buf, count, ncclChar, peer, comm, stream));
    ncclGroupEnd();

    return 0;
}

int experts_parallelism(hipStream_t stream, ncclComm_t comm, int rank, int count, void *send_buf, void *recv_buf) {

    int commFirstRank = 16*(rank / 16);
    int peer;

    ncclGroupStart();
    for (int off=0; off < 16; off++) {
        peer = commFirstRank + off;
        NCCLCHECK(ncclSend(send_buf, count, ncclChar, peer, comm, stream));
        NCCLCHECK(ncclRecv(recv_buf, count, ncclChar, peer, comm, stream));
    }
    ncclGroupEnd();
    return 0;
}

testResult_t InitData(struct threadArgs* args, ncclDataType_t type, ncclRedOp_t op, int root, int rep, int in_place) {
  size_t sendcount = args->sendBytes / wordSize(type);
  size_t recvcount = args->expectedBytes / wordSize(type);
  int nranks = args->nProcs*args->nThreads*args->nGpus;

  for (int i=0; i<args->nGpus; i++) {
    CUDACHECK(hipSetDevice(args->gpus[i]));
    int rank = ((args->proc*args->nThreads + args->thread)*args->nGpus + i);
    CUDACHECK(hipMemset(args->recvbuffs[i], 0, args->expectedBytes));
    void* data = in_place ? args->recvbuffs[i] : args->sendbuffs[i];
    TESTCHECK(InitData(data, sendcount, 0, type, ncclSum, 33*rep + rank, 1, 0));
    for (int j=0; j<nranks; j++) {
      size_t partcount = sendcount/nranks;
      TESTCHECK(InitData((char*)args->expected[i] + j*partcount*wordSize(type), partcount, rank*partcount, type, ncclSum, 33*rep + j, 1, 0));
    }
    CUDACHECK(hipDeviceSynchronize());
  }
  // We don't support in-place alltoall
  args->reportErrors = in_place ? 0 : 1;
  return testSuccess;
}

void GetBw(size_t count, int typesize, double sec, double* algBw, double* busBw, int nranks) { // TODO
  double baseBw = (double)(count * nranks * typesize) / 1.0E9 / sec;

  *algBw = baseBw;
  double factor = ((double)(nranks-1))/((double)(nranks));
  *busBw = baseBw * factor;
}

void GetCollByteCount(size_t *sendcount, size_t *recvcount, size_t *paramcount, size_t *sendInplaceOffset, size_t *recvInplaceOffset, size_t count, int nranks) {
    char *env;
    size_t experts_reduction_count, parallel_op_count;

    env = getenv("EXPERTS_REDUCTION_COUNT");
    experts_reduction_count = env ? (size_t) parsesize(env) : count;

    env = getenv("PARALLEL_OP_COUNT");
    parallel_op_count = env ? (size_t) parsesize(env) : count;

    count = (size_t) std::max(experts_reduction_count, parallel_op_count);

    *sendcount = (count/nranks)*nranks;
    *recvcount = (count/nranks)*nranks;
    *sendInplaceOffset = 0;
    *recvInplaceOffset = 0;
    *paramcount = count/nranks;
}

testResult_t RunColl(void* sendbuff, void* recvbuff, size_t count, ncclDataType_t type, ncclRedOp_t op, int root, ncclComm_t comm, hipStream_t stream, hipStream_t stream2) {
    /* The env var EXPERTS_REDUCTIONS_OP controls the collective simulating experts reductions
     * should be 0 for Reduce Scatter and 1 for All Gather
     * The env var PARALLEL_OP controls the additional coll, 0=nothing, 1=experts parallelism, 2=pipeline parallelism
     *
     * The count is the number of elements sent, therefore count * elemSize is the total size in bytes
     * For Allgather, the count is the number of element sent by each rank, thus total length of the vector is count * num_ranks 
     * For reduce scatter it is the number of elements received by the root, thus total length of the vector is count
     * For pipeline/expert parallelism it is the number of elements sent by each rank to each other rank
     *
     * Disclaimer: here we are sending ncclChar only, so elemSize is 1
     */
    char *env;
    int rank, size, experts_op, parallel_op;
    ncclResult_t state;
    size_t experts_reduction_count, parallel_op_count;
    NCCLCHECK(ncclCommCount(comm, &size));
    NCCLCHECK(ncclCommUserRank(comm, &rank));


    env = getenv("EXPERTS_REDUCTIONS_OP");
    experts_op = env ? atoi(env) : 0;

    env = getenv("PARALLEL_OP");
    parallel_op = env ? atoi(env) : 0;

    env = getenv("EXPERTS_REDUCTION_COUNT");
    experts_reduction_count = env ? (size_t) parsesize(env) : count;

    env = getenv("PARALLEL_OP_COUNT");
    parallel_op_count = env ? (size_t) parsesize(env) : count;

    if (size < 32){
    // if (size != num_ranks){
        printf("This test is meant to be ran with at least 32 ranks.");
        return testNcclError;
    }

    ncclComm_t expertsComm;
    NCCLCHECK(ncclCommSplit(comm, rank % 32, 0, &expertsComm, NULL));
    do {
        NCCLCHECK(ncclCommGetAsyncError(comm, &state));
    } while(state == ncclInProgress);

    switch (experts_op){
        case 0:
            NCCLCHECK(ncclReduceScatter((char *)sendbuff, (char *) recvbuff, experts_reduction_count, ncclChar, ncclSum, expertsComm, stream));
            break;
        case 1:
            NCCLCHECK(ncclAllGather((char *)sendbuff, (char *) recvbuff, experts_reduction_count, ncclChar, expertsComm, stream));
            break;
        default:
            printf("Invalid experts_op value, should be 0 for RS or 1 for AG, but received %d\n", experts_op);
            return testNcclError;
    }

    switch (parallel_op){
        case 0:
            break;
        case 1:
            experts_parallelism(stream2, comm, rank, parallel_op_count, sendbuff, recvbuff);
            break;
        case 2:
            pipeline_parallelism(stream2, comm, rank, parallel_op_count, sendbuff, recvbuff);
            break;
        default:
            printf("Invalid parallel op value, should be 0 for nothing, 1 for experts parallelism or 2 for pipeline parallelism");
            return testNcclError;
    }
    

    ncclCommDestroy(expertsComm);
    return testSuccess;
}


struct testColl moeBenchmarkTest = {
  "MoeBenchmark",
  GetCollByteCount,
  InitData,
  GetBw,
  RunColl,
};

void GetBuffSize(size_t *sendcount, size_t *recvcount, size_t count, int nranks) {
  size_t paramcount, sendInplaceOffset, recvInplaceOffset;
  GetCollByteCount(sendcount, recvcount, &paramcount, &sendInplaceOffset, &recvInplaceOffset, count, nranks);
}

testResult_t RunTest(struct threadArgs* args, int root, ncclDataType_t type, const char* typeName, ncclRedOp_t op, const char* opName) {
  args->collTest = &moeBenchmarkTest;
  ncclDataType_t *run_types;
  const char **run_typenames;
  int type_count;

  PRINT("EXPERTS_REDUCTION_COUNT=%s, PARALLEL_OP_COUNT=%s\n", getenv("EXPERTS_REDUCTION_COUNT"), getenv("PARALLEL_OP_COUNT"));

  if ((int)type != -1) {
    type_count = 1;
    run_types = &type;
    run_typenames = &typeName;
  } else {
    type_count = test_typenum;
    run_types = test_types;
    run_typenames = test_typenames;
  }

  for (int i=0; i<type_count; i++) {
      TESTCHECK(TimeTest(args, run_types[i], run_typenames[i], (ncclRedOp_t)0, "none", -1));
  }
  return testSuccess;
}

struct testEngine moeBenchmarkEngine = {
  GetBuffSize,
  RunTest
};

#pragma weak ncclTestEngine=moeBenchmarkEngine
